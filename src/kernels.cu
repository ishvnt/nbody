#include "hip/hip_runtime.h"
#include "main.h"
#include "kernels.cuh"

hipError_t check_error(hipError_t err)
{
    if(err != hipSuccess) printf("error: %s\n", hipGetErrorString(err));
    return err;
}

__global__
void init_galaxy(Point* points, int n, float radius, float centre_x, float centre_y)
{
    int idx = threadIdx.x + (blockDim.x * blockIdx.x);
    int stride = blockDim.x * gridDim.x;
    for(int i = idx; i < n; i += stride)
    {
        hiprandState state;
        hiprand_init(clock()+idx, 0, 0, &state);

        float theta = hiprand_uniform(&state) * 2 * M_PI;
        float phi = acosf(2.0f * hiprand_uniform(&state) - 1.0f); 
        float r = powf(hiprand_uniform(&state), 1.0f / 3.0f) * radius; 

        points[i].x = centre_x + ( r * sinf(phi) * cosf(theta) );
        
        points[i].y = centre_y + ( r * sinf(phi) * sinf(theta) );
        
        points[i].vx = sinf(theta)*20;
        points[i].vy = -cosf(theta)*20;
        points[i].ax = 0.00f;
        points[i].ay = 0.00f;
    }
}

__global__
void update_vel(Point* points, int n, float dt, float softening, float centre_x, float centre_y)
{
    int idx = threadIdx.x + (blockDim.x * blockIdx.x);
    int stride = blockDim.x * gridDim.x;
    for(int i = idx; i < n; i += stride)
    {
        float Fx = 0.00f, Fy = 0.00f;
        for(int j = 0; j < n; j++)
        {
            if(i == j)
            {
                float bh_m = 3e4;
                float bh_dx = points[i].x - centre_x;
                float bh_dy = points[i].y - centre_y;
                float bh_rinv = rsqrtf( (bh_dx * bh_dx) + (bh_dy * bh_dy) + softening );
                float bh_F = bh_m * bh_rinv * bh_rinv;
                Fx += bh_F * bh_dx * bh_rinv;
                Fy += bh_F * bh_dy * bh_rinv;
                continue;
            }
            float dx = points[i].x - points[j].x;
            float dy = points[i].y - points[j].y;
            float r_inv = rsqrtf( (dx * dx) + (dy * dy) + softening);
            float F = r_inv*r_inv;
            Fx +=  F * dx  * r_inv;
            Fy +=  F * dy  * r_inv;
        }
        points[i].vx -= Fx * dt * 0.5f;
        points[i].vy -= Fy * dt * 0.5f;
    }
}

__global__
void update_pos(Point* points, int n, float dt)
{
    int idx = threadIdx.x + (blockDim.x * blockIdx.x);
    int stride = blockDim.x * gridDim.x;
    for(int i = idx; i < n; i += stride)
    {
        points[i].x += points[i].vx * dt;
        points[i].y += points[i].vy * dt;
    }
}

__global__
void update_pos_verlet(Point* points, int n, float dt)
{
    int idx = threadIdx.x + (blockDim.x * blockIdx.x);
    int stride = blockDim.x * gridDim.x;
    for(int i = idx; i < n; i += stride)
    {
        points[i].x += ( points[i].vx * dt ) + ( 0.5f * points[i].ax * dt * dt );
        points[i].y += ( points[i].vy * dt ) + ( 0.5f * points[i].ay * dt * dt );
    }
}

__global__
void update_vel_verlet(Point* points, int n, float dt, float softening, float centre_x, float centre_y)
{
    int idx = threadIdx.x + (blockDim.x * blockIdx.x);
    int stride = blockDim.x * gridDim.x;
    for(int i = idx; i < n; i += stride)
    {
        float Fx = 0.00f, Fy = 0.00f;

        points[i].vx += 0.5f * points[i].ax * dt;
        points[i].vy += 0.5f * points[i].ay * dt;

        for(int j = 0; j < n; j++)
        {
            if(i == j)
            {
                float bh_m = 4e4;
                float bh_dx = points[i].x - centre_x;
                float bh_dy = points[i].y - centre_y;
                float bh_rinv = rsqrtf( (bh_dx * bh_dx) + (bh_dy * bh_dy) + softening );
                float bh_F = bh_m * bh_rinv * bh_rinv;
                Fx += bh_F * bh_dx * bh_rinv;
                Fy += bh_F * bh_dy * bh_rinv;
                continue;
            }
            float dx = points[i].x - points[j].x;
            float dy = points[i].y - points[j].y;
            float r_inv = rsqrtf( (dx * dx) + (dy * dy) + softening);
            float F = r_inv*r_inv;
            Fx +=  F * dx  * r_inv;
            Fy +=  F * dy  * r_inv;
        }

        points[i].ax = -Fx;
        points[i].ay = -Fy;

        points[i].vx += 0.5f * points[i].ax * dt;
        points[i].vy += 0.5f * points[i].ay * dt;
    }
}