#include "hip/hip_runtime.h"
#include <iostream>
#include <getopt.h>
#include "main.h"
#include "display.h"
#include "points.h"
#include "kernels.cuh"

int main(int argc, char *argv[])
{
    int dev_id;
    check_error( hipGetDevice(&dev_id) );

    int num_of_SM;
    check_error( hipDeviceGetAttribute(&num_of_SM, hipDeviceAttributeMultiprocessorCount, dev_id) );

    params_t* params;
    check_error( hipMallocManaged(&params, sizeof(params_t)) );
    // default parameters
    params->n = 1 << 15;
    params->dt = 0.005f;
    params->softening = 5.00f;
    params->radius = 300.00f;
    params->centre.x = 800.00f;
    params->centre.y = 400.00f;
    params->screen_width = 1600;
    params->screen_height = 800;
    params->dev_id = dev_id;
    params->thread_dim = {1024, 1, 1};
    params->block_dim = {(unsigned int)32 * num_of_SM, 1, 1};

    // change parameters according to command line arguments provided
    if (handle_args(argc, argv, params) == -1)
        return 1;

    Point* points;
    check_error( hipMallocManaged(&points, (params->n) * sizeof(Point)) );

    Display* win = new Display(params);

    // n/2 points in first galaxy
    init_galaxy<<<params->block_dim, params->thread_dim>>>(points, params->n/2, params->radius, params->centre);
    // n/2 points in second smaller galaxy
    init_galaxy<<<params->block_dim, params->thread_dim>>>(points + (params->n/2), params->n/2, 150.00f, {300.00f, 200.00f});
    check_error( hipGetLastError() );

    check_error( hipMemPrefetchAsync(params, sizeof(params_t), hipCpuDeviceId) );
    check_error( hipDeviceSynchronize() );

    win->loop(points, params);
    delete win;

    hipFree(points);
    hipFree(params);
    return 0;
}

int handle_args(int argc, char* argv[], params_t* params)
{
    int option;
    const char* optstring = "t:n:r:x:y:w:h:";

    while ((option = getopt(argc, argv, optstring)) != -1)
    {
        switch (option)
        {
        case 'n':
            params->n = 1 << atoi(optarg);
            break;
        case 't':
            params->dt = atof(optarg);
            break;
        case 'r':
            params->radius = atof(optarg);
            break;
        case 'x':
            params->centre.x = atof(optarg);
            break;
        case 'y':
            params->centre.y = atof(optarg);
            break;
        case 'w':
            params->screen_width = atoi(optarg);
            params->centre.x = params->screen_width * 0.5f;
            break;
        case 'h':
            params->screen_height = atoi(optarg);
            params->centre.y = params->screen_height * 0.5f;
            break;

        default:
            std::cout << "usage: ./nbody [ARGS]\n\
arguments: \n \
\t-n  number of bodies, calculated as pow(2, n), e.g if n = 10, then number of bodies = pow(2, 10) = 1024 \n \
\t-t  timestep (dt)\n \
\t-x  x-coordinate of centre of galaxy\n \
\t-y  y-coordinate of centre of galaxy\n \
\t-w  width of window\n \
\t-h  height of window\n";
            return -1;
        }
    }
    return 0;
}