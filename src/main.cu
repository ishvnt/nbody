#include "main.h"
#include "kernels.cuh"

int main()
{
    Point* points;
    check_error( hipMallocManaged(&points, n_points*sizeof(Point)) );

    int devId;
    check_error( hipGetDevice(&devId) );

    int num_of_SM;
    check_error( hipDeviceGetAttribute(&num_of_SM, hipDeviceAttributeMultiprocessorCount, devId) );

    hipMemPrefetchAsync(points, n_points*sizeof(Point), devId);

    int num_of_threads = 1024;
    int num_of_blocks = 32 * num_of_SM;

    init_points<<<num_of_blocks, num_of_threads>>>(points);

    check_error( hipGetLastError() );

    check_error( hipDeviceSynchronize() );

    Display* win = new Display();
    win->loop(points);
    delete win;

    hipFree(points);
    return 0;
}